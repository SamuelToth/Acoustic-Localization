#include "hip/hip_runtime.h"
#include "constants.cu"
#include "distances.cu"

#include<vector>
#include<iostream>
#include<stdio.h>


#define MAX_THREADS 1024
#define FREQUENCY_RANGE 4


/**findMatches
  *Finds all the matching wave pairs between two input sources.
  *Output does not filter unlikely results and false posatives
  *
  *batch1: array containing the FftResults from one input device
  *batch1Count: the number of elements in batch1
  *batch2: array containing the FftResults from one input device
  *batch2Count: the number of elements in batch2
  *matchMatrix 1D bool array representing a 2d bool matrix
  */

//find matches sequentially
void findMatchesSeq(FftResult* batch1, 
                 unsigned int batch1Count, 
                 FftResult* batch2,
                 unsigned int batch2Count,
                 bool* matchMatrix) {
  
  //Find all matching waves between two sources
//  FftResult wave1 = batch1[absoluteIndex];
  for (unsigned int i = 0; i < batch2Count; i++) {
    //find the waves are within an acceptable range there is a match
    for (unsigned int k=0; k<batch1Count; k++) {
      if (abs(batch1[k].frequency - batch2[i].frequency) <= FREQUENCY_RANGE){
        //create wavePair
        matchMatrix[i + k] = true;
        //printf("i: %i j: %i\n", absoluteIndex, i);
      }
      else { //else not a match
        matchMatrix[i + k] = false;
      }
    }
  }
  return;
}

__global__
void findMatches(FftResult* batch1, 
                 unsigned int batch1Count, 
                 FftResult* batch2,
                 unsigned int batch2Count,
                 bool* matchMatrix) {
                 
  //get threads position and return early if out of bounds
  int absoluteIndex = blockIdx.x *blockDim.x + threadIdx.x;  
  if (absoluteIndex > batch1Count) {
    return;
  }
  
  //init matchMatrix to false
  for (unsigned int i = 0; i < batch2Count; i++) {
    matchMatrix[absoluteIndex * batch1Count + i] = false;
  }
  __syncthreads();
  
  //Find all matching waves between two sources
  FftResult wave1 = batch1[absoluteIndex];
  for (unsigned int i = 0; i < batch2Count; i++) {
    //find the waves are within an acceptable range there is a match
    if (abs(wave1.frequency - batch2[i].frequency) <= FREQUENCY_RANGE){
      //create wavePair
      matchMatrix[absoluteIndex * batch1Count + i] = true;
      //printf("i: %i j: %i\n", absoluteIndex, i);
    }
  }
}

__global__
void findMatches2d(FftResult* batch1, 
                 unsigned int batch1Count, 
                 FftResult* batch2,
                 unsigned int batch2Count,
                 bool* matchMatrix) {
                 
  int absoluteIndex = blockIdx.x * blockDim.x * blockDim.y
      + threadIdx.y * blockDim.x + threadIdx.x;
      
  //init matchMatrix to false
  //for (unsigned int i = 0; i < batch2Count; i++) {
    matchMatrix[absoluteIndex] = false;
  //}
  __syncthreads();
  
  int crossBlockXIndex = blockIdx.x * blockDim.x + threadIdx.x;
  int crossBlockYIndex = blockIdx.y * blockDim.y + threadIdx.y;
  
  if (abs(batch1[crossBlockXIndex].frequency - batch2[crossBlockYIndex].frequency) <= FREQUENCY_RANGE) {
    matchMatrix[absoluteIndex] = true;
  }
  
}


WaveMatches findAllMatches(FftBatch* batches, unsigned int batchCount) {
  
  //store matrix for each batch compare
  WaveMatches matches;  
  
  for (unsigned int i = 0; i < batchCount; i++) {
    for (unsigned int j = i + 1; j < batchCount; j++) {
      //create return matrix for findMatches kernal on device
      bool* d_matchMatrix;  
      hipMalloc(&d_matchMatrix, sizeof(bool) * batches[i].size * batches[j].size);
      
      //for better performance make the larger batch the first batch in the kernal call
      FftBatch bigBatch;
      FftBatch littleBatch;
      
      if (batches[i].size > batches[j].size) {
        bigBatch = batches[i];
        littleBatch = batches[j];
      } else {
        bigBatch = batches[j];
        littleBatch = batches[i];
      }
      
      bool* h_matchMatrix = (bool *)malloc(bigBatch.size * littleBatch.size * sizeof(bool));

      //Move Fft results to kernal
      FftResult* d_batch1;
      FftResult* d_batch2;
      hipMalloc(&d_batch1, sizeof(FftResult) * bigBatch.size);
      hipMalloc(&d_batch2, sizeof(FftResult) * littleBatch.size);
      
      hipMemcpy(d_batch1, bigBatch.fftResults, 
                 sizeof(FftResult) * bigBatch.size, hipMemcpyHostToDevice);
      hipMemcpy(d_batch2, littleBatch.fftResults, 
                 sizeof(FftResult) * littleBatch.size, hipMemcpyHostToDevice);
      
      //call find matches kernal
      int threads = bigBatch.size % MAX_THREADS;
      int blocks = bigBatch.size / MAX_THREADS + 1;
      dim3 bDim(bigBatch.size, littleBatch.size, 0);
      //findMatches2d<<<threads, bDim>>>(d_batch1, bigBatch.size, 
                                       //d_batch2, littleBatch.size, 
                                       //d_matchMatrix);
      findMatchesSeq(bigBatch.fftResults, bigBatch.size, littleBatch.fftResults,littleBatch.size, h_matchMatrix);
      
      //copy matchMatrix to host and store in return vector
      //bool* matchMatrix = (bool*)malloc(sizeof(bool) * bigBatch.size * littleBatch.size);
      
      //hipMemcpy(matchMatrix, d_matchMatrix, sizeof(bool) * bigBatch.size *littleBatch.size,hipMemcpyDeviceToHost);
                 
      matches.matches.push_back(h_matchMatrix);
      matches.widths.push_back(bigBatch.size);
      matches.heights.push_back(littleBatch.size);
       
            
      //free memory
      hipFree(d_batch1);
      hipFree(d_batch2);
      hipFree(d_matchMatrix);
            
    }
  }

  return matches;
}

int main(){

  FftBatch batch1;
  FftResult results1[5];
  FftBatch batch2;
  FftResult results2[3];
  FftBatch batch3;
  FftResult results3[2];
  
  for (unsigned int i = 0; i < 5; i++) {
    FftResult result;
    result.frequency = i;
    result.offset = i * 2;
    results1[i] = result;
  }
  batch1.fftResults = results1;
  batch1.size = 5;
  
  for (unsigned int i = 0; i < 3; i++) {
    FftResult result;
    result.frequency = i + 1;
    result.offset = i * 4;
    results2[i] = result;
  }
  batch2.fftResults = results2;
  batch2.size = 3;
  
  for (unsigned int i = 0; i < 2; i++) {
    FftResult result;
    result.frequency = i + 3;
    result.offset = i;
    results3[i] = result;
  }
  batch3.fftResults = results3;
  batch3.size = 2;
  
  FftBatch batches[3];
  batches[0] = batch1;
  batches[1] = batch2;
  batches[2] = batch3;
  
  WaveMatches matches = findAllMatches(batches, 3);
  

  std::cout << "matches: " << matches.matches.size() << std::endl;
  for (unsigned int i = 0; i < matches.matches.size(); i++) {
    //std::cout << "width: " << matches.widths[i] << std::endl;
    //std::cout << "height: " << matches.heights[i] << std::endl; 
    
    //std::cout << matches.matches[0][1][0] << std::endl;
    
    
    for (unsigned int j = 0; j < matches.widths[i]; j++) {
      for (unsigned int k = 0; k < matches.heights[i]; k++) {
        if (matches.matches[matches.widths[i] * j + k]){
          std::cout<<"("<<j<<","<<k<<")"<<std::endl;
        }
      
      }
    }
  }


  return 0;
}
