//#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <vector>

typedef float decimal;

struct Distances {
  decimal* distances;
};

struct Coordinate {
  decimal x;
  decimal y;
};

struct FftResult {
  decimal frequency;
  decimal offset;
};

struct FftBatch {
  FftResult* fftResults;
  unsigned int size;  
};

struct WavePair {
	int waveIdx1;
	int waveIdx2;
	decimal offset;
};


struct WavePairContainer{
	int firstFFT;
	int secondFFT;
	WavePair* wavePairArray;
	int wavePairCount;
	//thrust::device_vector<WavePair> wavePairArray;
};

struct WaveMatches {
  std::vector<bool*> matches;
  std::vector<unsigned int> widths;
  std::vector<unsigned int> heights;
  std::vector<int> widthBatches;
  std::vector<int> heightBatches;
};

struct GpuWaveMatches {
  bool** matches;
  unsigned int matchesCount;
  
  unsigned int* widths;
  unsigned int widthsCount;
  
  unsigned int* heights;
  unsigned int heightsCount;
  
  int* widthBatches;
  unsigned int widthBatchesCount;
  
  int* heightBatches;
  unsigned int heightBatchesCount;
  
};

void freeGpuWaveMatches(GpuWaveMatches* gpuMatches)
{
  for (unsigned int i = 0; i < gpuMatches->matchesCount; i++)
  {
    hipFree(gpuMatches->matches[i]);
  }
  hipFree(gpuMatches->widths);
  hipFree(gpuMatches->heights);
  hipFree(gpuMatches->widthBatches);
  hipFree(gpuMatches->heightBatches);
}

void WaveMatchesToGpu(const WaveMatches& matches, GpuWaveMatches* gpuMatches)
{
  //allocate memory for the GpuWaveMatches struct
  hipMalloc(&gpuMatches, sizeof(GpuWaveMatches));
  
  //copy the the matches array and all match matrix
  bool** gpuMatchesArray;
  hipMalloc(&gpuMatchesArray, sizeof(bool*) * matches.matches.size());
  for (unsigned int i = 0; i < matches.matches.size(); i++)
  {
    bool* gpuMatchMatrix;
    hipMalloc(&gpuMatchMatrix, sizeof(bool) * matches.widths[i] * matches.heights[i]);
    hipMemcpy(gpuMatchMatrix, matches.matches[i], sizeof(bool) * matches.widths[i] * matches.heights[i], hipMemcpyHostToDevice);
    gpuMatchesArray[i] = gpuMatchMatrix;
  }
  gpuMatches->matches = gpuMatchesArray;
  gpuMatches->matchesCount = matches.matches.size();
  
  //copy the stored widths
  unsigned int* gpuWidths;
  hipMalloc(&gpuWidths, sizeof(unsigned int) * matches.widths.size());
  hipMemcpy(gpuWidths, &matches.widths[0], sizeof(unsigned int) * matches.widths.size(), hipMemcpyHostToDevice);
  gpuMatches->widths = gpuWidths;
  gpuMatches->widthsCount = matches.widths.size();
  
  //copy the stored heights
  unsigned int* gpuHeights;
  hipMalloc(&gpuHeights, sizeof(unsigned int) * matches.heights.size());
  hipMemcpy(gpuHeights, &matches.heights[0], sizeof(unsigned int) * matches.heights.size(), hipMemcpyHostToDevice);
  gpuMatches->heights = gpuHeights;
  gpuMatches->heightsCount = matches.heights.size();
  
  //copy stored widthBatches
  int* gpuWidthBatches;
  hipMalloc(&gpuWidthBatches, sizeof(int) * matches.widthBatches.size());
  hipMemcpy(gpuWidthBatches, &matches.widthBatches[0], sizeof(int) * matches.widthBatches.size(), hipMemcpyHostToDevice);
  gpuMatches->widthBatches = gpuWidthBatches;
  gpuMatches->widthBatchesCount = matches.widthBatches.size();
  
  //copy stored heightBatches
  int* gpuHeightBatches;
  hipMalloc(&gpuHeightBatches, sizeof(int) * matches.heightBatches.size());
  hipMemcpy(gpuHeightBatches, &matches.heightBatches[0], sizeof(int) * matches.heightBatches.size(), hipMemcpyHostToDevice);
  gpuMatches->heightBatches = gpuHeightBatches;
  gpuMatches->heightBatchesCount = matches.heightBatches.size();
}




