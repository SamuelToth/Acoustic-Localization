#ifndef __lcl_constants
#define __lcl_constants

//#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

typedef float decimal;

struct MicData {
  hipfftDoubleComplex** micData;
  int* waveLengths;
  int numberOfBatches;
};

struct Distances {
  decimal* distances;
};

struct Coordinate {
  decimal x;
  decimal y;
};

struct FftResult {
  decimal frequency;
  decimal offset;
};

struct FftBatch {
  FftResult* fftResults;
  unsigned int size;  
};

struct WavePair {
	int waveIdx1;
	int waveIdx2;
	decimal offset;
};


struct WavePairContainer{
	int firstFFT;
	int secondFFT;
	WavePair* wavePairArray;
	int wavePairCount;
	//thrust::device_vector<WavePair> wavePairArray;
};

struct WaveMatches {
  std::vector<bool*> matches;
  std::vector<unsigned int> widths;
  std::vector<unsigned int> heights;
  std::vector<int> widthBatches;
  std::vector<int> heightBatches;
};

struct GpuWaveMatches {
  bool** matches;
  unsigned int matchesCount;
  
  unsigned int* widths;
  unsigned int widthsCount;
  
  unsigned int* heights;
  unsigned int heightsCount;
  
  int* widthBatches;
  unsigned int widthBatchesCount;
  
  int* heightBatches;
  unsigned int heightBatchesCount;
  
};

void GpuWaveMatchesToHost(GpuWaveMatches* h_gpuWaveMatches, GpuWaveMatches* d_gpuWaveMatches)
{
  h_gpuWaveMatches = (GpuWaveMatches*)malloc(sizeof(GpuWaveMatches));
  hipMemcpy(h_gpuWaveMatches, d_gpuWaveMatches, sizeof(GpuWaveMatches), hipMemcpyDeviceToHost);
  
  unsigned int* tempWidths = (unsigned int*)malloc( sizeof(unsigned int) * h_gpuWaveMatches->widthsCount);
  hipMemcpy(tempWidths, h_gpuWaveMatches->widths, sizeof(unsigned int) * h_gpuWaveMatches->widthsCount, hipMemcpyDeviceToHost);
  h_gpuWaveMatches->widths = tempWidths;
  unsigned int* tempHeights = (unsigned int*)malloc( sizeof(unsigned int) * h_gpuWaveMatches->heightsCount);
  hipMemcpy(tempHeights, h_gpuWaveMatches->heights, sizeof(unsigned int) * h_gpuWaveMatches->heightsCount, hipMemcpyDeviceToHost);
  h_gpuWaveMatches->heights = tempHeights;
  int* tempWidthBatches = (int*)malloc( sizeof(int) * h_gpuWaveMatches->widthBatchesCount);
  hipMemcpy(tempWidthBatches, h_gpuWaveMatches->widthBatches, sizeof(int) * h_gpuWaveMatches->widthBatchesCount, hipMemcpyDeviceToHost);
  h_gpuWaveMatches->widthBatches = tempWidthBatches;
  int* tempHeightBatches = (int*)malloc( sizeof(int) * h_gpuWaveMatches->heightBatchesCount);
  hipMemcpy(tempHeightBatches, h_gpuWaveMatches->heightBatches, sizeof(int) * h_gpuWaveMatches->heightBatchesCount, hipMemcpyDeviceToHost);
  h_gpuWaveMatches->heightBatches = tempHeightBatches;
  
  //copy matches
  hipMemcpy(h_gpuWaveMatches->matches, d_gpuWaveMatches->matches, sizeof(bool*) * h_gpuWaveMatches->matchesCount, hipMemcpyDeviceToHost);
  for (unsigned int i = 0; i < h_gpuWaveMatches->matchesCount; i++){
    bool* tempMatches = (bool*)malloc( sizeof(bool) * h_gpuWaveMatches->widths[i] * h_gpuWaveMatches->heights[i] );
    hipMemcpy(tempMatches, h_gpuWaveMatches->matches[i], sizeof(bool) * h_gpuWaveMatches->widths[i] * h_gpuWaveMatches->heights[i], hipMemcpyDeviceToHost);
    h_gpuWaveMatches->matches[i] = tempMatches;
  }
}

void freeGpuWaveMatches(GpuWaveMatches* gpuMatches)
{
  GpuWaveMatches* h_gpuMatches = (GpuWaveMatches*)malloc(sizeof(GpuWaveMatches));
  hipMemcpy(h_gpuMatches, gpuMatches, sizeof(GpuWaveMatches), hipMemcpyDeviceToHost);
  
  for (unsigned int i = 0; i < h_gpuMatches->matchesCount; i++)
  {
    hipFree(h_gpuMatches->matches[i]);
  }
  hipFree(h_gpuMatches->widths);
  hipFree(h_gpuMatches->heights);
  hipFree(h_gpuMatches->widthBatches);
  hipFree(h_gpuMatches->heightBatches);
  
  free(h_gpuMatches);
  hipFree(gpuMatches);
}

void WaveMatchesToGpu(const WaveMatches& matches, GpuWaveMatches* gpuMatches)
{
  //allocate memory for the GpuWaveMatches struct
  hipMalloc(&gpuMatches, sizeof(GpuWaveMatches));
  
  //copy the the matches array and all match matrix
  bool** gpuMatchesArray;
  hipMalloc(&gpuMatchesArray, sizeof(bool*) * matches.matches.size());
  
  for (unsigned int i = 0; i < matches.matches.size(); i++)
  {
    bool* gpuMatchMatrix;
    hipMalloc(&gpuMatchMatrix, sizeof(bool) * matches.widths[i] * matches.heights[i]);
    hipMemcpy(gpuMatchMatrix, matches.matches[i], sizeof(bool) * matches.widths[i] * matches.heights[i], hipMemcpyHostToDevice);
    gpuMatchesArray[i] = gpuMatchMatrix;
  }
  gpuMatches->matches = gpuMatchesArray;
  gpuMatches->matchesCount = matches.matches.size();
  
  //copy the stored widths
  unsigned int* gpuWidths;
  hipMalloc(&gpuWidths, sizeof(unsigned int) * matches.widths.size());
  hipMemcpy(gpuWidths, &matches.widths[0], sizeof(unsigned int) * matches.widths.size(), hipMemcpyHostToDevice);
  gpuMatches->widths = gpuWidths;
  gpuMatches->widthsCount = matches.widths.size();
  
  //copy the stored heights
  unsigned int* gpuHeights;
  hipMalloc(&gpuHeights, sizeof(unsigned int) * matches.heights.size());
  hipMemcpy(gpuHeights, &matches.heights[0], sizeof(unsigned int) * matches.heights.size(), hipMemcpyHostToDevice);
  gpuMatches->heights = gpuHeights;
  gpuMatches->heightsCount = matches.heights.size();
  
  //copy stored widthBatches
  int* gpuWidthBatches;
  hipMalloc(&gpuWidthBatches, sizeof(int) * matches.widthBatches.size());
  hipMemcpy(gpuWidthBatches, &matches.widthBatches[0], sizeof(int) * matches.widthBatches.size(), hipMemcpyHostToDevice);
  gpuMatches->widthBatches = gpuWidthBatches;
  gpuMatches->widthBatchesCount = matches.widthBatches.size();
  
  //copy stored heightBatches
  int* gpuHeightBatches;
  hipMalloc(&gpuHeightBatches, sizeof(int) * matches.heightBatches.size());
  hipMemcpy(gpuHeightBatches, &matches.heightBatches[0], sizeof(int) * matches.heightBatches.size(), hipMemcpyHostToDevice);
  gpuMatches->heightBatches = gpuHeightBatches;
  gpuMatches->heightBatchesCount = matches.heightBatches.size();
}
#endif



