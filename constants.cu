#ifndef __lcl_constants
#define __lcl_constants

//#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <stdio.h>

typedef float decimal;

struct MicData {
  hipfftDoubleComplex** micData;
  int* waveLengths;
  int numberOfBatches;
};

struct Distances {
  decimal* distances;
};

struct Coordinate {
  decimal x;
  decimal y;
};

struct FftResult {
  decimal frequency;
  decimal offset;
};

struct FftBatch {
  FftResult* fftResults;
  unsigned int size;  
};

struct WavePair {
	int waveIdx1;
	int waveIdx2;
	decimal offset;
};


struct WavePairContainer{
	int firstFFT;
	int secondFFT;
	WavePair* wavePairArray;
	int wavePairCount;
	//thrust::device_vector<WavePair> wavePairArray;
};

struct WaveMatches {
  std::vector<bool*> matches;
  std::vector<unsigned int> widths;
  std::vector<unsigned int> heights;
  std::vector<int> widthBatches;
  std::vector<int> heightBatches;
};

struct GpuWaveMatches {
  bool** matches;
  unsigned int matchesCount;
  
  unsigned int* widths;
  unsigned int widthsCount;
  
  unsigned int* heights;
  unsigned int heightsCount;
  
  int* widthBatches;
  unsigned int widthBatchesCount;
  
  int* heightBatches;
  unsigned int heightBatchesCount;
  
};

void GpuWaveMatchesToHost(GpuWaveMatches* h_gpuWaveMatches, GpuWaveMatches* d_gpuWaveMatches)
{
  h_gpuWaveMatches = (GpuWaveMatches*)malloc(sizeof(GpuWaveMatches));
  hipMemcpy(h_gpuWaveMatches, d_gpuWaveMatches, sizeof(GpuWaveMatches), hipMemcpyDeviceToHost);
  printf("gwmth 1\r\n"); fflush(NULL);
  unsigned int* tempWidths = (unsigned int*)malloc( sizeof(unsigned int) * h_gpuWaveMatches->widthsCount);
  hipMemcpy(tempWidths, &h_gpuWaveMatches->widths, sizeof(unsigned int) * h_gpuWaveMatches->widthsCount, hipMemcpyDeviceToHost);
  h_gpuWaveMatches->widths = tempWidths;
  printf("gwmth 2\r\n"); fflush(NULL);
  unsigned int* tempHeights = (unsigned int*)malloc( sizeof(unsigned int) * h_gpuWaveMatches->heightsCount);
  hipMemcpy(tempHeights, &h_gpuWaveMatches->heights, sizeof(unsigned int) * h_gpuWaveMatches->heightsCount, hipMemcpyDeviceToHost);
  printf("gwmth 3\r\n"); fflush(NULL);
  h_gpuWaveMatches->heights = tempHeights;
  int* tempWidthBatches = (int*)malloc( sizeof(int) * h_gpuWaveMatches->widthBatchesCount);
  hipMemcpy(tempWidthBatches, &h_gpuWaveMatches->widthBatches, sizeof(int) * h_gpuWaveMatches->widthBatchesCount, hipMemcpyDeviceToHost);
  h_gpuWaveMatches->widthBatches = tempWidthBatches;
  printf("gwmth 4\r\n"); fflush(NULL);
  int* tempHeightBatches = (int*)malloc( sizeof(int) * h_gpuWaveMatches->heightBatchesCount);
  hipMemcpy(tempHeightBatches, &h_gpuWaveMatches->heightBatches, sizeof(int) * h_gpuWaveMatches->heightBatchesCount, hipMemcpyDeviceToHost);
  h_gpuWaveMatches->heightBatches = tempHeightBatches;
  
  //copy matches
  hipMemcpy(&h_gpuWaveMatches->matches, &d_gpuWaveMatches->matches, sizeof(bool*) * h_gpuWaveMatches->matchesCount, hipMemcpyDeviceToHost);
  for (unsigned int i = 0; i < h_gpuWaveMatches->matchesCount; i++){
    printf("widths: %i , heights: %i\r\n", h_gpuWaveMatches->widths[i], h_gpuWaveMatches->heights[i]);
    bool* tempMatches = (bool*)malloc( sizeof(bool) * h_gpuWaveMatches->widths[i] * h_gpuWaveMatches->heights[i] );
    printf("brianIs");
    hipMemcpy(tempMatches, &h_gpuWaveMatches->matches[i], sizeof(bool) * h_gpuWaveMatches->widths[i] * h_gpuWaveMatches->heights[i], hipMemcpyDeviceToHost);
    printf("notReal");
    h_gpuWaveMatches->matches[i] = tempMatches;
    printf("its a thing here\r\n");fflush(NULL);
  }
}

void freeGpuWaveMatches(GpuWaveMatches* gpuMatches)
{
  GpuWaveMatches* h_gpuMatches = (GpuWaveMatches*)malloc(sizeof(GpuWaveMatches));
  hipMemcpy(h_gpuMatches, gpuMatches, sizeof(GpuWaveMatches), hipMemcpyDeviceToHost);
  
  for (unsigned int i = 0; i < h_gpuMatches->matchesCount; i++)
  {
    hipFree(&h_gpuMatches->matches[i]);
  }
  printf("stuff n things\r\n"); fflush(NULL);
  hipFree(&h_gpuMatches->widths);
  hipFree(&h_gpuMatches->heights);
  hipFree(&h_gpuMatches->widthBatches);
  hipFree(&h_gpuMatches->heightBatches);
  
  free(h_gpuMatches);
  hipFree(gpuMatches);
}

void WaveMatchesToGpu(const WaveMatches& matches, GpuWaveMatches* gpuMatches)
{
  //allocate memory for the GpuWaveMatches struct
  hipMalloc(&gpuMatches, sizeof(GpuWaveMatches));
  GpuWaveMatches* h_gpuMatches = (GpuWaveMatches*)malloc(sizeof(GpuWaveMatches));
  
  //copy the the matches array and all match matrix
  bool** gpuMatchesArray;
  hipMalloc(&gpuMatchesArray, sizeof(bool*) * matches.matches.size());
  bool** h_gpuMatchesArray = (bool**)malloc(sizeof(bool*) * matches.matches.size());
  
  
  for (unsigned int i = 0; i < matches.matches.size(); i++)
  {
    bool* gpuMatchMatrix;
    hipMalloc(&gpuMatchMatrix, sizeof(bool) * matches.widths[i] * matches.heights[i]);
    hipMemcpy(gpuMatchMatrix, matches.matches[i], sizeof(bool) * matches.widths[i] * matches.heights[i], hipMemcpyHostToDevice);
    h_gpuMatchesArray[i] = gpuMatchMatrix;
  }
  
  hipMemcpy(gpuMatchesArray, h_gpuMatchesArray, sizeof(bool*) * matches.matches.size(), hipMemcpyHostToDevice);
  h_gpuMatches->matches = gpuMatchesArray;
  h_gpuMatches->matchesCount = matches.matches.size();
  
  //copy the stored widths
  unsigned int* gpuWidths;
  hipMalloc(&gpuWidths, sizeof(unsigned int) * matches.widths.size());
  hipMemcpy(gpuWidths, &matches.widths[0], sizeof(unsigned int) * matches.widths.size(), hipMemcpyHostToDevice);
  h_gpuMatches->widths = gpuWidths;
  h_gpuMatches->widthsCount = matches.widths.size();
  
  //copy the stored heights
  unsigned int* gpuHeights;
  hipMalloc(&gpuHeights, sizeof(unsigned int) * matches.heights.size());
  hipMemcpy(gpuHeights, &matches.heights[0], sizeof(unsigned int) * matches.heights.size(), hipMemcpyHostToDevice);
  h_gpuMatches->heights = gpuHeights;
  h_gpuMatches->heightsCount = matches.heights.size();
  
  //copy stored widthBatches
  int* gpuWidthBatches;
  hipMalloc(&gpuWidthBatches, sizeof(int) * matches.widthBatches.size());
  hipMemcpy(gpuWidthBatches, &matches.widthBatches[0], sizeof(int) * matches.widthBatches.size(), hipMemcpyHostToDevice);
  h_gpuMatches->widthBatches = gpuWidthBatches;
  h_gpuMatches->widthBatchesCount = matches.widthBatches.size();
  
  //copy stored heightBatches
  int* gpuHeightBatches;
  hipMalloc(&gpuHeightBatches, sizeof(int) * matches.heightBatches.size());
  hipMemcpy(gpuHeightBatches, &matches.heightBatches[0], sizeof(int) * matches.heightBatches.size(), hipMemcpyHostToDevice);
  h_gpuMatches->heightBatches = gpuHeightBatches;
  h_gpuMatches->heightBatchesCount = matches.heightBatches.size();
  
  
  hipMemcpy(gpuMatches, h_gpuMatches, sizeof(GpuWaveMatches), hipMemcpyHostToDevice);
  //TODO: free host memory;
}
#endif



