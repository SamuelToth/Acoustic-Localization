#ifndef __lcl_constants
#define __lcl_constants

//#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

typedef float decimal;

struct MicData {
  hipfftDoubleComplex** micData;
  int* waveLengths;
  int numberOfBatches;
};

struct Distances {
  decimal* distances;
};

struct Coordinate {
  decimal x;
  decimal y;
};

struct FftResult {
  decimal frequency;
  decimal offset;
};

struct FftBatch {
  FftResult* fftResults;
  unsigned int size;  
};

struct WavePair {
	int waveIdx1;
	int waveIdx2;
	decimal offset;
};


struct WavePairContainer{
	int firstFFT;
	int secondFFT;
	WavePair* wavePairArray;
	int wavePairCount;
	//thrust::device_vector<WavePair> wavePairArray;
};

struct WaveMatches {
  std::vector<bool*> matches;
  std::vector<unsigned int> widths;
  std::vector<unsigned int> heights;
  std::vector<int> widthBatches;
  std::vector<int> heightBatches;
};

struct GpuWaveMatches {
  bool** matches;
  unsigned int matchesCount;
  
  unsigned int* widths;
  unsigned int widthsCount;
  
  unsigned int* heights;
  unsigned int heightsCount;
  
  int* widthBatches;
  unsigned int widthBatchesCount;
  
  int* heightBatches;
  unsigned int heightBatchesCount;
  
};

void GpuWaveMatchesToHost(GpuWaveMatches* h_gpuWaveMatches, GpuWaveMatches* d_gpuWaveMatches)
{
  h_gpuWaveMatches = (GpuWaveMatches*)malloc(sizeof(GpuWaveMatches));
  hipMemcpy(h_gpuWaveMatches, d_gpuWaveMatches, sizeof(GpuWaveMatches), hipMemcpyDeviceToHost);
  
  hipMemcpy(h_gpuWaveMatches->widths, d_gpuWaveMatches->widths, sizeof(unsigned int) * h_gpuWaveMatches->widthsCount, hipMemcpyDeviceToHost);
  hipMemcpy(h_gpuWaveMatches->heights, d_gpuWaveMatches->heights, sizeof(unsigned int) * h_gpuWaveMatches->heightsCount, hipMemcpyDeviceToHost);
  hipMemcpy(h_gpuWaveMatches->widthBatches, d_gpuWaveMatches->widthBatches, sizeof(int) * h_gpuWaveMatches->widthBatchesCount, hipMemcpyDeviceToHost);
  hipMemcpy(h_gpuWaveMatches->heightBatches, d_gpuWaveMatches->heightBatches, sizeof(int) * h_gpuWaveMatches->heightBatchesCount, hipMemcpyDeviceToHost);
  
  //copy matches
  hipMemcpy(h_gpuWaveMatches->matches, d_gpuWaveMatches->matches, sizeof(bool*) * h_gpuWaveMatches->matchesCount, hipMemcpyDeviceToHost);
  for (unsigned int i = 0; i < h_gpuWaveMatches->matchesCount; i++){
    hipMemcpy(h_gpuWaveMatches->matches[i], d_gpuWaveMatches->matches[i], sizeof(bool) * h_gpuWaveMatches->widths[i] * h_gpuWaveMatches->heights[i], hipMemcpyDeviceToHost);
  }
}

void freeGpuWaveMatches(GpuWaveMatches* gpuMatches)
{
  for (unsigned int i = 0; i < gpuMatches->matchesCount; i++)
  {
    hipFree(gpuMatches->matches[i]);
  }
  hipFree(gpuMatches->widths);
  hipFree(gpuMatches->heights);
  hipFree(gpuMatches->widthBatches);
  hipFree(gpuMatches->heightBatches);
}

void WaveMatchesToGpu(const WaveMatches& matches, GpuWaveMatches* gpuMatches)
{
  //allocate memory for the GpuWaveMatches struct
  hipMalloc(&gpuMatches, sizeof(GpuWaveMatches));
  
  //copy the the matches array and all match matrix
  bool** gpuMatchesArray;
  hipMalloc(&gpuMatchesArray, sizeof(bool*) * matches.matches.size());
  for (unsigned int i = 0; i < matches.matches.size(); i++)
  {
    bool* gpuMatchMatrix;
    hipMalloc(&gpuMatchMatrix, sizeof(bool) * matches.widths[i] * matches.heights[i]);
    hipMemcpy(gpuMatchMatrix, matches.matches[i], sizeof(bool) * matches.widths[i] * matches.heights[i], hipMemcpyHostToDevice);
    gpuMatchesArray[i] = gpuMatchMatrix;
  }
  gpuMatches->matches = gpuMatchesArray;
  gpuMatches->matchesCount = matches.matches.size();
  
  //copy the stored widths
  unsigned int* gpuWidths;
  hipMalloc(&gpuWidths, sizeof(unsigned int) * matches.widths.size());
  hipMemcpy(gpuWidths, &matches.widths[0], sizeof(unsigned int) * matches.widths.size(), hipMemcpyHostToDevice);
  gpuMatches->widths = gpuWidths;
  gpuMatches->widthsCount = matches.widths.size();
  
  //copy the stored heights
  unsigned int* gpuHeights;
  hipMalloc(&gpuHeights, sizeof(unsigned int) * matches.heights.size());
  hipMemcpy(gpuHeights, &matches.heights[0], sizeof(unsigned int) * matches.heights.size(), hipMemcpyHostToDevice);
  gpuMatches->heights = gpuHeights;
  gpuMatches->heightsCount = matches.heights.size();
  
  //copy stored widthBatches
  int* gpuWidthBatches;
  hipMalloc(&gpuWidthBatches, sizeof(int) * matches.widthBatches.size());
  hipMemcpy(gpuWidthBatches, &matches.widthBatches[0], sizeof(int) * matches.widthBatches.size(), hipMemcpyHostToDevice);
  gpuMatches->widthBatches = gpuWidthBatches;
  gpuMatches->widthBatchesCount = matches.widthBatches.size();
  
  //copy stored heightBatches
  int* gpuHeightBatches;
  hipMalloc(&gpuHeightBatches, sizeof(int) * matches.heightBatches.size());
  hipMemcpy(gpuHeightBatches, &matches.heightBatches[0], sizeof(int) * matches.heightBatches.size(), hipMemcpyHostToDevice);
  gpuMatches->heightBatches = gpuHeightBatches;
  gpuMatches->heightBatchesCount = matches.heightBatches.size();
}
#endif



