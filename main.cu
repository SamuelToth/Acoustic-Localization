#ifndef __lcl_main
#define __lcl_main

#define NX 512//256
#define BATCH 1

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hipfft/hipfftw.h>
#include <cmath>

#include <iostream>

int handleCufft(double freq)
{
  //std::cout << "===========input:==========="<<std::endl;
  //gen data
  hipfftDoubleComplex* h_data = (hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)*NX);
  for (unsigned int i = 0; i < NX; i++){
    h_data[i].x = sin( 2 * M_PI * freq * (double)i / NX);
    h_data[i].y = 0.0;
    //std::cout<<i<<": "<<h_data[i].x<< "+i * " << h_data[i].y <<std::endl;
    //std::cout<<h_data[i].x<<std::endl;
  }

  //std::cout<<std::endl;
  //std::cin.get();
  
  hipfftHandle plan;// = cufftCreate();
  hipfftDoubleComplex *data;
  hipfftDoubleComplex *outData;
  hipMalloc((void**)&data, sizeof(hipfftDoubleComplex)*(NX)*BATCH);               //example code: cudaMalloc((void**)&data, sizeof(cufftComplex)*(NX/2+1)*BATCH);  
  hipMemcpy(data, h_data, sizeof(hipfftDoubleComplex)*NX*BATCH, hipMemcpyHostToDevice);
  hipMalloc((void**)&outData, sizeof(hipfftDoubleComplex)*(NX / 2 + 1)*BATCH);
  

  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to allocate\n"); 
    return 1;
  }

  if (hipfftPlan1d(&plan, NX, HIPFFT_D2Z, BATCH) != HIPFFT_SUCCESS){ 
    fprintf(stderr, "CUFFT error: Plan creation failed"); 
    return 1;	
  }

	/* Use the CUFFT plan to transform the signal in place. */ 
  if (hipfftExecD2Z(plan, (hipfftDoubleReal*)data, outData) != HIPFFT_SUCCESS){ 
    fprintf(stderr, "CUFFT error: ExecC2C Forward failed"); 
    return 1;	
  }

  if (hipDeviceSynchronize() != hipSuccess){ 
    fprintf(stderr, "Cuda error: Failed to synchronize\n"); 
    return 1;
  }
  
  hipMemcpy(h_data, outData, sizeof(hipfftDoubleComplex)*(NX / 2 + 1)*BATCH, hipMemcpyDeviceToHost);
  
  //std::cin.get();

  //std::cout<<"~~~~~~~output~~~~~~~"<<std::endl;
   for (unsigned int i = 0; i < NX / 2 + 1; i++) {
    //std::cout << i << ": " <<h_data[i].x << "+ i*" << h_data[i].y << std::endl;
    std::cout << h_data[i].x << std::endl;
  }
  
  std::cin.get();
  std::cin.get();
  
  for (unsigned int i = 0; i < NX / 2 + 1; i++) {
    std::cout << i << ": " <<h_data[i].x << " + i*" << h_data[i].y << std::endl;
    //std::cout << h_data[i].x << std::endl;
  }
  
  hipfftDestroy(plan); 
  hipFree(data);
  hipFree(outData);
  
  free(h_data);
  return 0;
}


int main() {

  double freq;
  std::cout << "freq: ";
  std::cin >> freq;
  
  handleCufft(freq);

  
  return 0;
}

#endif
