#include "hip/hip_runtime.h"
#ifndef __lcl_fftWork
#define __lcl_fftWork

#define NX 512//256
#define BATCH 1

#include "constants.cu"

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hipfft/hipfftw.h>
#include <cmath>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */

#include <iostream>

#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>


__global__
void getFftResults(int oldArraySize, int* goodIndexes, bool* d_validFrequencies, FftResult* fftResults, hipfftDoubleComplex* rawCufftResults)
{
  int thid = blockIdx.x *blockDim.x + threadIdx.x;
  if (thid >= oldArraySize)
  {
    return;
  }
  
  if(d_validFrequencies[thid])
  {
    fftResults[goodIndexes[thid]].frequency = thid;
    fftResults[goodIndexes[thid]].offset = rawCufftResults[thid].y;
    
  }
}


__global__
void trueIfGreater(bool* results, const double* const fftReals, unsigned int size, double flagValue)
{
  int thid = blockIdx.x *blockDim.x + threadIdx.x;
  if (thid >= size)
  {
    return;
  }
  
  results[thid] = (abs(fftReals[thid]) > flagValue);
}


int getFftBatch(FftBatch* batch, hipfftDoubleComplex* h_data)
{
  hipfftHandle plan;// = hipfftCreate();
  hipfftDoubleComplex *data;
  hipfftDoubleComplex *outData;
  hipMalloc((void**)&data, sizeof(hipfftDoubleComplex)*(NX)*BATCH);               //example code: hipMalloc((void**)&data, sizeof(hipfftComplex)*(NX/2+1)*BATCH);  
  hipMemcpy(data, h_data, sizeof(hipfftDoubleComplex)*NX*BATCH, hipMemcpyHostToDevice);
  hipMalloc((void**)&outData, sizeof(hipfftDoubleComplex)*(NX / 2 + 1)*BATCH);
  

  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to allocate\n"); 
    return 1;
  }

  if (hipfftPlan1d(&plan, NX, HIPFFT_D2Z, BATCH) != HIPFFT_SUCCESS){ 
    fprintf(stderr, "CUFFT error: Plan creation failed"); 
    return 1;	
  }

	/* Use the CUFFT plan to transform the signal in place. */ 
  if (hipfftExecD2Z(plan, (hipfftDoubleReal*)data, outData) != HIPFFT_SUCCESS){ 
    fprintf(stderr, "CUFFT error: ExecC2C Forward failed"); 
    return 1;	
  }

  if (hipDeviceSynchronize() != hipSuccess){ 
    fprintf(stderr, "Cuda error: Failed to synchronize\n"); 
    return 1;
  }
  
  int outputSize = (NX / 2 + 1)*BATCH;
  
  //hipMemcpy(h_data, outData, sizeof(hipfftDoubleComplex)*(NX / 2 + 1)*BATCH, hipMemcpyDeviceToHost);
  thrust::host_vector<hipfftDoubleComplex> rawFft(outData, outData + outputSize);
  

  thrust::host_vector<double> fftReals(outputSize);
  //double* fftReals = (double*)malloc(sizeof(double) * outputSize);
 
  double rawSum = 0;
  for (unsigned int i = 0; i < outputSize; i++)
  {
    rawSum += rawFft[i].x;
    fftReals[i] = rawFft[i].x;
  }
  double rawAverage = rawSum / outputSize;
  
  thrust::device_vector<double> d_fftReals(fftReals);
  //thrust::device_vector<double> d_validReals(outputSize);
  
  bool* d_validFrequencies;
  hipMalloc(&d_validFrequencies, sizeof(bool) * outputSize);
  

  
  
  int blockSizeInt = 1024;
  int gridSizeInt = outputSize / 1024 + 1;
  
  trueIfGreater<<<gridSizeInt, blockSizeInt>>>(d_validFrequencies, thrust::raw_pointer_cast(d_fftReals.data()), outputSize, rawAverage * 5);
  
  int* goodIndexes = (int*)malloc(sizeof(int) * outputSize);
  thrust::exclusive_scan(d_validFrequencies, d_validFrequencies + outputSize, goodIndexes);
  int goodVals = thrust::reduce(d_validFrequencies, d_validFrequencies + outputSize);
  
  FftResult* d_fftResults;
  hipMalloc(&d_fftResults, sizeof(FftResult) * goodVals);
  
  batch->size = goodVals;
  batch->fftResults = (FftResult*)malloc(sizeof(FftResult) * goodVals);
  
  int* d_goodIndexes;
  hipMalloc(&d_goodIndexes, sizeof(int) * outputSize);
  hipMemcpy(d_goodIndexes, goodIndexes, sizeof(int) * outputSize, hipMemcpyHostToDevice);
  
  getFftResults<<<gridSizeInt, blockSizeInt>>>(outputSize, d_goodIndexes,d_validFrequencies, d_fftResults, outData);
  
  hipMemcpy(batch->fftResults, d_fftResults, sizeof(FftResult) * goodVals, hipMemcpyDeviceToHost);
  
  hipFree(d_fftResults);
  //void getFftResults<<<gridSizeInt, blockSizeInt>>>(outputSize, goodIndexes, d_validFrequencies, d_fftResults, outData);
  
  //bool* h_validFrequencies = (bool*)malloc(sizeof(bool) * outputSize);
  //hipMemcpy(h_validFrequencies, d_validFrequencies, sizeof(bool) * outputSize, hipMemcpyDeviceToHost);
  
  /*for (unsigned int i = 0 i < outputSize; i++)
  {
    if (h_validFrequencies[i])
    {
      
    }
  }*/
  
  hipFree(d_goodIndexes);
  hipfftDestroy(plan); 
  hipFree(data);
  hipFree(outData);
  
  return 0;
}

#endif
