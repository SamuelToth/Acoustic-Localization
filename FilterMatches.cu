#include "hip/hip_runtime.h"
#include "constants.cu"

#include<vector>
#include<iostream>
#include<stdio.h>


#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/scan.h>

 
__global__
void buildHistogramForTriples(const GpuWaveMatches* allMatches,
                              unsigned int** matchHistograms)
{
  // 3D block 1D grid
  int matchIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int widthIdx = blockIdx.x * blockDim.y + threadIdx.y;
  int heightIdx = blockIdx.x * blockDim.z + threadIdx.z;
  
  //return if index is bad
  if (matchIdx >= allMatches->matchesCount)
  {
    return;
  }
  
  if (widthIdx >= allMatches->widths[matchIdx] 
        || heightIdx >= allMatches->heights[matchIdx])
  {
    return;
  }
  
  //find matrix and batch information from allMatches
  unsigned int flatMatrixPosition = allMatches->widths[matchIdx] * widthIdx + heightIdx;
  int widthBatchNum = allMatches->widthBatches[matchIdx];
  int heightBatchNum = allMatches->heightBatches[matchIdx];
  unsigned int matrixVal = (unsigned int)allMatches->matches[matchIdx][flatMatrixPosition];
  
  
  //set histogram values
  atomicAdd(&matchHistograms[widthBatchNum][widthIdx], matrixVal);
  atomicAdd(&matchHistograms[heightBatchNum][heightIdx], matrixVal);
}


__global__
void removeNonTripleMatches(GpuWaveMatches* allMatches,
                            const unsigned int * const * const matchHistograms)
{
  // 3D block 1D grid
  int matchIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int widthIdx = blockIdx.x * blockDim.y + threadIdx.y;
  int heightIdx = blockIdx.x * blockDim.z + threadIdx.z;
  
  //return if index is bad
  if (matchIdx >= allMatches->matchesCount)
  {
    return;
  }
  
  if (widthIdx >= allMatches->widths[matchIdx] 
        || heightIdx >= allMatches->heights[matchIdx])
  {
    return;
  }
  
  //find matrix and batch information from allMatches
  unsigned int flatMatrixPosition = allMatches->widths[matchIdx] * widthIdx + heightIdx;
  int widthBatchNum = allMatches->widthBatches[matchIdx];
  int heightBatchNum = allMatches->heightBatches[matchIdx]; 
  
  //set value based on histogram
  if (matchHistograms[widthBatchNum][widthIdx] < 3 || matchHistograms[heightBatchNum][heightIdx] < 3)
  {
    allMatches->matches[matchIdx][flatMatrixPosition] = 0;
  }
}



/*filterForTriples
 *Removes all matches with frequencies that aren't found across
 * at least three results
 *
 *allMatches: WaveMatches struct containing all the match matricies for each
 * FftBatch
 *
 *MatchHistograms: an array of histograms (one for each FftBatch) for tracking
 * the number of matches for each frequency
 */
 void filterForTriples(WaveMatches& allMatches,
                      unsigned int** matchHistograms,
                      GpuWaveMatches* d_outMatches)
{
  //move waveMatches to teh gpu
  GpuWaveMatches* gpuWaveMatches = NULL;
  WaveMatchesToGpu(allMatches, gpuWaveMatches);
  
  //determine kernel dimentions
  const int maxThreadsPerBlock = 512;
  unsigned int* maxWidth = thrust::max_element(&allMatches.widths[0], &allMatches.widths[0] + allMatches.widths.size());
  unsigned int* maxHeight = thrust::max_element(&allMatches.heights[0], &allMatches.heights[0] + allMatches.heights.size());
  double widthHeightRatio = *maxWidth / *maxHeight;
  unsigned int blockSizeIntX = allMatches.matches.size() % maxThreadsPerBlock;
  unsigned int blockSizeIntY = (maxThreadsPerBlock - blockSizeIntX) * widthHeightRatio;
  unsigned int blockSizeIntZ = blockSizeIntY;
  if (widthHeightRatio > 1)
  {
    blockSizeIntY = (maxThreadsPerBlock - blockSizeIntX) / widthHeightRatio;
  }
  else
  {
    blockSizeIntY = (maxThreadsPerBlock - blockSizeIntX) * widthHeightRatio;
  }
  blockSizeIntZ = (maxThreadsPerBlock - blockSizeIntX) - blockSizeIntY;
  unsigned int gridSizeInt = (*maxWidth * *maxHeight * allMatches.matches.size()) / (blockSizeIntY * blockSizeIntX * blockSizeIntZ) + 1;
  dim3 blockSize(blockSizeIntX, blockSizeIntY, blockSizeIntZ);
  dim3 gridSize(gridSizeInt);
  
  //fill histograms
  buildHistogramForTriples<<<gridSize, blockSize>>>(gpuWaveMatches,matchHistograms);
  
  //remove frequencies that dont match at least three times
  removeNonTripleMatches<<<gridSize, blockSize>>>(gpuWaveMatches, matchHistograms);
  
  d_outMatches = gpuWaveMatches;
}




__global__
void matrixToWavePair(bool* d_waveMatches,
                      const unsigned int* const outputPositions,
                      unsigned int matrixSize,
                      unsigned int matrixWidth,
                      unsigned int matrixHeight,
                      WavePair* d_wavePairs,
                      unsigned int pairCount)
{
  int thid = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
  int thidX = blockIdx.x *blockDim.x + threadIdx.x;
  int thidY = blockIdx.x *blockDim.y + threadIdx.y;
  if (thid >= matrixSize || thidX >= matrixWidth || thidY >= matrixHeight)
  {
    return;
  }
  
  unsigned int position = matrixWidth * thidX + thidY;
  if (d_waveMatches[position])
  {
    //TODO; ensure that waveidx1 will always be the width of the matrix
    d_wavePairs[outputPositions[position]].waveIdx1 = thidX;
    d_wavePairs[outputPositions[position]].waveIdx2 = thidY;
  }
}




void findWavePairs(FftBatch* batches,
              unsigned int batchCount,
              GpuWaveMatches* d_waveMatches
              WavePairContainer* wpContainers)
{
  GpuWaveMatches* h_waveMatches;
  GpuWaveMatchesToHost(h_waveMatches, d_waveMatches);
  
  for (unsigned int i = 0; i < h_waveMatches->matchesCount i++)
  {
    //determine the number of wavePairs and their positions in the output array
    unsigned int matrixSize = waveMatches->widths[i] * waveMatches->heights[i];
    bool* scanResult = (bool*)malloc(sizeof(bool) * matrixSize);
    thrust::exclusive_scan(h_waveMatches->matches[i], h_waveMatches->matches[i] + matrixSize, scanResult);
    unsigned int total = scanResult[matrixSize - 1] + h_waveMatches->matches[i][matrixSize - 1];
    
    //create wavePairContainer
    wpContainers[i].wavePairCount = total;
    wpContainers[i].firstFFT = h_waveMatches.widthBatches[i];
    wpContainers[i].secondFFT = h_waveMatches.heightBatches[i];
    wpContainers[i].wavePairArray = (WavePair*)malloc(sizeof(WavePair) * total);
    
    //populate teh wavePairArray
    WavePair* d_wavePairs;
    hipMalloc(&d_wavePairs, sizeof(WavePair) * total);
    
    

    free(scanResult);
    hipFree(d_wavePairs);
    
  }
  //TODO: free h_waveMatches;
}


/*filterMatches: removes all invalid matches. Returns valid matches in wave pair
 * containers
 *
 *batches: array of Fft data taken from input mics. Each batch is data
 * from one mic.
 *
 *batchCount: the number of batches
 *
 *allMatches: Raw match data
 *
 *wavePairContainers: output. Pairs of waves from different mics
 * with the same frequency
 *
 *containerCount: the number of wavePairContainers to output
 */
void filterMatches(FftBatch* batches,
                   unsigned int batchCount,
                   WaveMatches* allMatches,
                   WavePairContainer* wavePairContainers,
                   unsigned int containerCount)
{
  //Create a histogram for each WaveMatch width
  unsigned int** d_matchHistograms;
  hipMalloc(&d_matchHistograms, sizeof(unsigned int*) * batchCount);
  for (unsigned int i = 0; i < batchCount; i++)
  {
    unsigned int* d_matchHistogram;
    hipMalloc(&d_matchHistogram, sizeof(unsigned int) * batches[i].size);
    hipMemset(d_matchHistogram, 0, sizeof(unsigned int) * batches[i].size);
    d_matchHistograms[i] = d_matchHistogram;
  }
  
  GpuWaveMatches* d_waveMatches;
  filterForTriples(*allMatches, d_matchHistograms, d_waveMatches);
  
  

  //free device waveMatches
  freeGpuWaveMatches(d_waveMatches);
  
  //free histogram memory
  for (unsigned int i = 0; i < batchCount; i++)
  {
    hipFree(d_matchHistograms[i]);
  }
  hipFree(d_matchHistograms);
}


int main()
{

}
